#include "hip/hip_runtime.h"
#include "Tunnel.h"

Tunnel::Tunnel(std::string object_file, uint width, uint height, float dx, float scale, uint threads, uint gs_iters, float speed) : Fluid(width, height, dx) {

    threads = threads;
    gs_iters = gs_iters;

    this -> speed = speed;

    sf::Image object;
    object.loadFromFile(object_file);

    object_mask = new uint8_t[width * height];

    for(int i = 0; i < width; i++){
        for(int j = 0; j < height; j++){

            
            float object_x = object.getSize().x/2 + ((i - (float) width/2) / scale);
            float object_y = object.getSize().y/2 + ((j - (float) height/2) / scale);

            if(object_x >= 0 && object_x < object.getSize().x && object_y >= 0 && object_y < object.getSize().y){
                object_mask[coords2index(i, j, width)] = object.getPixel((int) object_x, (int) object_y).a;
            } else {
                object_mask[coords2index(i, j, width)] = 0;
            }
        }
    }

    uint8_t *d_object_mask = NULL;

    hipMalloc(&d_object_mask, width * height);
    hipMemcpy(d_object_mask, object_mask, width * height, hipMemcpyHostToDevice);

    object_mask_CUDA = d_object_mask;
}

Tunnel::~Tunnel(){
    delete object_mask;
    hipFree(object_mask_CUDA);
}



void Tunnel::draw_object(sf::RenderWindow& window, float block_size){

    sf::RectangleShape rect(sf::Vector2f(block_size, block_size));

    sf::Color c(255, 255, 255);

    for(uint i = 0; i < width; i++){

        for(uint j = 0; j < height; j++){

            rect.setPosition(i * block_size, j * block_size);
            c.a = object_mask[coords2index(i, j, width)];
            rect.setFillColor(c);
            window.draw(rect);
        }

    }

}


void Tunnel::set_boundaries_sector(Particle* particles, uint start, uint end, uint identifier){

}

__global__
void set_boundaries_kernel(Particle* particles, uint width, uint height, uint identifier, float speed, uint8_t *object_mask){

    uint i = blockIdx.x;
    uint j = threadIdx.x;


    uint smoke_start = (uint) (0.40 * height);
    uint smoke_end = (uint) (0.60 * height);

    if((i == 0 || i == width - 1) && (j >= 1 && j < height - 1)){


        if(identifier == 1){
            particles[coords2index(0, j, width)].vx = speed;
            particles[coords2index(width - 1, j, width)].vx = particles[coords2index(width - 2, j, width)].vx;
        }

        if(identifier == 2){
            particles[coords2index(0, j, width)].vy = particles[coords2index(1, j, width)].vy;
            particles[coords2index(width - 1, j, width)].vy = particles[coords2index(width - 2, j, width)].vy;
        }

        if(identifier == 3){
            particles[coords2index(0, j, width)].div = particles[coords2index(1, j, width)].div;
            particles[coords2index(width - 1, j, width)].div = particles[coords2index(width - 2, j, width)].div;
        }

        if(identifier == 4){
            particles[coords2index(0, j, width)].p = particles[coords2index(1, j, width)].p;
            particles[coords2index(width - 1, j, width)].p = particles[coords2index(width - 2, j, width)].p;
        }

        if(identifier == 5){
            particles[coords2index(0, j, width)].smoke = 0;
            particles[coords2index(width - 1, j, width)].smoke = particles[coords2index(width - 2, j, width)].smoke;
        
            if(j >= smoke_start && j <= smoke_end){
                particles[coords2index(0, j, width)].smoke = 0.5;
            }

        }
    }

    if(j == 0 || j == height - 1) {

        if(identifier == 1){
            particles[coords2index(i, 0, width)].vx = speed;
            particles[coords2index(i, height - 1, width)].vx = speed;
        }
        
        if(identifier == 2){
            particles[coords2index(i, 0, width)].vy = -particles[coords2index(i, 1, width)].vy;
            particles[coords2index(i, height - 1, width)].vy = -particles[coords2index(i, height - 2, width)].vy;
        }
        
        if(identifier == 3){
            particles[coords2index(i, 0, width)].div = particles[coords2index(i, 1, width)].div;
            particles[coords2index(i, height - 1, width)].div = particles[coords2index(i, height - 2, width)].div;
        }
        
        if(identifier == 4){
            particles[coords2index(i, 0, width)].p = particles[coords2index(i, 1, width)].p;
            particles[coords2index(i, height - 1, width)].p = particles[coords2index(i, height - 2, width)].p;        
        }
        
        if(identifier == 5){
            particles[coords2index(i, 0, width)].smoke = particles[coords2index(i, 1, width)].smoke;
            particles[coords2index(i, height - 1, width)].smoke = particles[coords2index(i, height - 2, width)].smoke;
        }
    }


    if(i > 0 && i < width - 1 && j > 0 && j < height - 1){

        particles[coords2index(i, j, width)].Fy = 9.81;


        Particle& p = particles[coords2index(i, j, width)];

        const uint8_t c = object_mask[coords2index(i, j, width)];         

        if(c == 255){
            if(identifier == 1)
                p.vx = 0;
            if(identifier == 2)
                p.vy = 0;
            if(identifier == 3)
                p.div = 0;
            if(identifier == 4)
                p.p = 0;
            if(identifier == 5)
                p.smoke = 0;

            uint count = 0;

            int neighbours[][2] = {{0, 1}, {1, 1}, {1, 0}, {1, -1}, {0, -1}, {-1, -1}, {-1, 0}, {-1, 1}};

            for(int k = 0; k < 8; k++){

                float sx = neighbours[k][0];
                float sy = neighbours[k][1];

                const uint8_t c1 = object_mask[coords2index(i + sx, j + sy, width)];
                Particle& n = particles[coords2index(i + sx, j + sy, width)];


                if(c1 != 255){

                    count ++;

                    float len_sqr = sx*sx + sy*sy;

                    float vx = ((n.vx*sx + n.vy*sy) / len_sqr) * sx;
                    float vy = ((n.vx*sx + n.vy*sy) / len_sqr) * (+sy);

                    if(identifier == 1)
                        p.vx += -vx;
                    
                    if(identifier == 2)
                        p.vy += -vy;

                    if(identifier == 3)
                        p.div += n.div;

                    if(identifier == 4)
                        p.p += n.p;

                    if(identifier == 5)
                        p.smoke += n.smoke;
                }
            
            }
            

            if(count > 0){
                if(identifier == 1)
                    p.vx = p.vx;
                
                if(identifier == 2)
                    p.vy = p.vy;

                if(identifier == 3)
                    p.div = p.div / count;
                
                if(identifier == 4)
                    p.p = p.p / count;

                if(identifier == 5)
                    p.smoke = p.smoke / count;
            }
        }
    }




}


void Tunnel::set_boundaries(Particle* particles, uint width, uint height, uint identifier){  
    set_boundaries_kernel<<<width, height>>>(particles, width, height, identifier, speed, object_mask_CUDA);
}


float Tunnel::calculate_lift(){
    float lift = 0;

    for(uint i = 0; i < width; i++){
        for(uint j = 0; j < height; j++){
            Particle& p = particles[coords2index(i, j, width)];

            const uint8_t c = object_mask[coords2index(i, j, width)];
            if(c == 255){
                lift += p.vy;
            }

        }
    }
       
    return lift;
}

float Tunnel::calculate_drag(){
    float drag = 0;

    for(uint i = 0; i < width; i++){
        for(uint j = 0; j < height; j++){
            Particle& p = particles[coords2index(i, j, width)];

            const uint8_t c = object_mask[coords2index(i, j, width)];
            if(c == 255){
                drag += -p.vx;
            }

        }
    }
       
    return drag;
}